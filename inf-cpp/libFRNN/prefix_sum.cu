#include "hip/hip_runtime.h"
// small change by Lixin to fit pytorch's API

/*
	Matt Dean - 1422434 - mxd434
	
	Goals implemented:
		- Block scan for arbitrary length small vectors - 'blockscan' function
		- Full scan for arbitrary length large vectors	- 'scan' function
			This function decides whether to perform a small (one block) scan or a full (n-level) scan depending on the length of the input vector
		- BCAO for both scans

	Hardware:
		CPU - Intel Core i5-4670k @ 3.4GHz
		GPU - NVIDIA GeForce GTX 760

	Timings:
		10,000,000 Elements
		  host     : 20749 ms
		  gpu      : 7.860768 ms
		  gpu bcao : 4.304064 ms
		
		For more results please see the comment at the bottom of this file

	Extra work:
		Due to the recursive nature of the full scan it can handle n > 3 levels 
	
*/

#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include "prefix_sum.h"

// scan.cuh
void sequential_scan(int* output, int* input, int length);
void blockscan(int *output, int *input, int length, bool bcao);
void scan(int *output, int *input, int length, bool bcao);

void scanLargeDeviceArray(int *output, int *input, int length, bool bcao);
void scanSmallDeviceArray(int *d_out, int *d_in, int length, bool bcao);
void scanLargeEvenDeviceArray(int *output, int *input, int length, bool bcao);


// kernels.cuh
__global__ void prescan_arbitrary(int *output, int *input, int n, int powerOfTwo);
__global__ void prescan_arbitrary_unoptimized(int *output, int *input, int n, int powerOfTwo);

__global__ void prescan_large(int *output, int *input, int n, int* sums);
__global__ void prescan_large_unoptimized(int *output, int *input, int n, int *sums);

__global__ void add(int *output, int length, int *n1);
__global__ void add(int *output, int length, int *n1, int *n2);


// utils.h
void _checkCudaError(const char *message, hipError_t err, const char *caller);
void printResult(const char* prefix, int result, long nanoseconds);
void printResult(const char* prefix, int result, float milliseconds);

bool isPowerOfTwo(int x);
int nextPowerOfTwo(int x);

long get_nanos();

/*///////////////////////////////////*/
/*            New API                */
/*///////////////////////////////////*/

void PrefixSumCUDA(
    const at::Tensor grid_cnt,
    int num_grids,
    at::Tensor grid_off) {
  
  scan(
    grid_off.contiguous().data_ptr<int>(),
    grid_cnt.contiguous().data_ptr<int>(),
    num_grids,
    true
  );

  return;
}

void PrefixSumCPU(
    const at::Tensor grid_cnt,
    int num_grids,
    at::Tensor grid_off) {

  sequential_scan(
    grid_off.contiguous().data_ptr<int>(),
    grid_cnt.contiguous().data_ptr<int>(),
    num_grids
  );

  return;
}


/*///////////////////////////////////*/
/*            scan.cu                */
/*///////////////////////////////////*/
#define checkCudaError(o, l) _checkCudaError(o, l, __func__)

int THREADS_PER_BLOCK = 512;
int ELEMENTS_PER_BLOCK = THREADS_PER_BLOCK * 2;

void sequential_scan(int* output, int* input, int length) {

	output[0] = 0; // since this is a prescan, not a scan
	for (int j = 1; j < length; ++j)
	{
		output[j] = input[j - 1] + output[j - 1];
	}

	return;
}

void blockscan(int *d_out, int *d_in, int length, bool bcao) {
	int powerOfTwo = nextPowerOfTwo(length);
	if (bcao) {
		prescan_arbitrary<<<1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int)>>>(d_out, d_in, length, powerOfTwo);
	}
	else {
		prescan_arbitrary_unoptimized<<<1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int)>>>(d_out, d_in, length, powerOfTwo);
  }
  
  return;
}

void scan(int *d_out, int *d_in, int length, bool bcao) {
	if (length > ELEMENTS_PER_BLOCK) {
		scanLargeDeviceArray(d_out, d_in, length, bcao);
	}
	else {
		scanSmallDeviceArray(d_out, d_in, length, bcao);
	}

	return;
}


void scanLargeDeviceArray(int *d_out, int *d_in, int length, bool bcao) {
	int remainder = length % (ELEMENTS_PER_BLOCK);
	if (remainder == 0) {
		scanLargeEvenDeviceArray(d_out, d_in, length, bcao);
	}
	else {
		// perform a large scan on a compatible multiple of elements
		int lengthMultiple = length - remainder;
		scanLargeEvenDeviceArray(d_out, d_in, lengthMultiple, bcao);

		// scan the remaining elements and add the (inclusive) last element of the large scan to this
		int *startOfOutputArray = &(d_out[lengthMultiple]);
		scanSmallDeviceArray(startOfOutputArray, &(d_in[lengthMultiple]), remainder, bcao);

		add<<<1, remainder>>>(startOfOutputArray, remainder, &(d_in[lengthMultiple - 1]), &(d_out[lengthMultiple - 1]));
	}
}

void scanSmallDeviceArray(int *d_out, int *d_in, int length, bool bcao) {
	int powerOfTwo = nextPowerOfTwo(length);

	if (bcao) {
		prescan_arbitrary<<<1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int)>>>(d_out, d_in, length, powerOfTwo);
	}
	else {
		prescan_arbitrary_unoptimized<<<1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int)>>>(d_out, d_in, length, powerOfTwo);
	}
}

void scanLargeEvenDeviceArray(int *d_out, int *d_in, int length, bool bcao) {
	const int blocks = length / ELEMENTS_PER_BLOCK;
	const int sharedMemArraySize = ELEMENTS_PER_BLOCK * sizeof(int);

	int *d_sums, *d_incr;
	hipMalloc((void **)&d_sums, blocks * sizeof(int));
	hipMalloc((void **)&d_incr, blocks * sizeof(int));

	if (bcao) {
		prescan_large<<<blocks, THREADS_PER_BLOCK, 2 * sharedMemArraySize>>>(d_out, d_in, ELEMENTS_PER_BLOCK, d_sums);
	}
	else {
		prescan_large_unoptimized<<<blocks, THREADS_PER_BLOCK, 2 * sharedMemArraySize>>>(d_out, d_in, ELEMENTS_PER_BLOCK, d_sums);
	}

	const int sumsArrThreadsNeeded = (blocks + 1) / 2;
	if (sumsArrThreadsNeeded > THREADS_PER_BLOCK) {
		// perform a large scan on the sums arr
		scanLargeDeviceArray(d_incr, d_sums, blocks, bcao);
	}
	else {
		// only need one block to scan sums arr so can use small scan
		scanSmallDeviceArray(d_incr, d_sums, blocks, bcao);
	}

	add<<<blocks, ELEMENTS_PER_BLOCK>>>(d_out, ELEMENTS_PER_BLOCK, d_incr);

	hipFree(d_sums);
	hipFree(d_incr);
}



/*///////////////////////////////////*/
/*            kernels.cu             */
/*///////////////////////////////////*/
#define SHARED_MEMORY_BANKS 32
#define LOG_MEM_BANKS 5

// There were two BCAO optimisations in the paper - this one is fastest
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_MEM_BANKS)

__global__ void prescan_arbitrary(int *output, int *input, int n, int powerOfTwo)
{
	extern __shared__ int temp[];// allocated on invocation
	int threadID = threadIdx.x;

	int ai = threadID;
	int bi = threadID + (n / 2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);


	if (threadID < n) {
		temp[ai + bankOffsetA] = input[ai];
		temp[bi + bankOffsetB] = input[bi];
	}
	else {
		temp[ai + bankOffsetA] = 0;
		temp[bi + bankOffsetB] = 0;
	}


	int offset = 1;
	for (int d = powerOfTwo >> 1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if (threadID == 0) {
		temp[powerOfTwo - 1 + CONFLICT_FREE_OFFSET(powerOfTwo - 1)] = 0; // clear the last element
	}

	for (int d = 1; d < powerOfTwo; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	if (threadID < n) {
		output[ai] = temp[ai + bankOffsetA];
		output[bi] = temp[bi + bankOffsetB];
	}
}

__global__ void prescan_arbitrary_unoptimized(int *output, int *input, int n, int powerOfTwo) {
	extern __shared__ int temp[];// allocated on invocation
	int threadID = threadIdx.x;

	if (threadID < n) {
		temp[2 * threadID] = input[2 * threadID]; // load input into shared memory
		temp[2 * threadID + 1] = input[2 * threadID + 1];
	}
	else {
		temp[2 * threadID] = 0;
		temp[2 * threadID + 1] = 0;
	}


	int offset = 1;
	for (int d = powerOfTwo >> 1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if (threadID == 0) { temp[powerOfTwo - 1] = 0; } // clear the last element

	for (int d = 1; d < powerOfTwo; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	if (threadID < n) {
		output[2 * threadID] = temp[2 * threadID]; // write results to device memory
		output[2 * threadID + 1] = temp[2 * threadID + 1];
	}
}


__global__ void prescan_large(int *output, int *input, int n, int *sums) {
	extern __shared__ int temp[];

	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * n;

	int ai = threadID;
	int bi = threadID + (n / 2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
	temp[ai + bankOffsetA] = input[blockOffset + ai];
	temp[bi + bankOffsetB] = input[blockOffset + bi];

	int offset = 1;
	for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}
	__syncthreads();


	if (threadID == 0) {
		sums[blockID] = temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)];
		temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;
	}

	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	output[blockOffset + ai] = temp[ai + bankOffsetA];
	output[blockOffset + bi] = temp[bi + bankOffsetB];
}

__global__ void prescan_large_unoptimized(int *output, int *input, int n, int *sums) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * n;

	extern __shared__ int temp[];
	temp[2 * threadID] = input[blockOffset + (2 * threadID)];
	temp[2 * threadID + 1] = input[blockOffset + (2 * threadID) + 1];

	int offset = 1;
	for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}
	__syncthreads();


	if (threadID == 0) {
		sums[blockID] = temp[n - 1];
		temp[n - 1] = 0;
	}

	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	output[blockOffset + (2 * threadID)] = temp[2 * threadID];
	output[blockOffset + (2 * threadID) + 1] = temp[2 * threadID + 1];
}


__global__ void add(int *output, int length, int *n) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * length;

	output[blockOffset + threadID] += n[blockID];
}

__global__ void add(int *output, int length, int *n1, int *n2) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * length;

	output[blockOffset + threadID] += n1[blockID] + n2[blockID];
}


/*///////////////////////////////////*/
/*            utils.cpp              */
/*///////////////////////////////////*/
void _checkCudaError(const char *message, hipError_t err, const char *caller) {
	if (err != hipSuccess) {
		fprintf(stderr, "Error in: %s\n", caller);
		fprintf(stderr, message);
		fprintf(stderr, ": %s\n", hipGetErrorString(err));
		exit(0);
	}
}

void printResult(const char* prefix, int result, long nanoseconds) {
	printf("  ");
	printf(prefix);
	printf(" : %i in %ld ms \n", result, nanoseconds / 1000);
}

void printResult(const char* prefix, int result, float milliseconds) {
	printf("  ");
	printf(prefix);
	printf(" : %i in %f ms \n", result, milliseconds);
}


// from https://stackoverflow.com/a/3638454
bool isPowerOfTwo(int x) {
	return x && !(x & (x - 1));
}

// from https://stackoverflow.com/a/12506181
int nextPowerOfTwo(int x) {
	int power = 1;
	while (power < x) {
		power *= 2;
	}
	return power;
}


// from https://stackoverflow.com/a/36095407
// Get the current time in nanoseconds
long get_nanos() {
	struct timespec ts;
	timespec_get(&ts, TIME_UTC);
	return (long)ts.tv_sec * 1000000000L + ts.tv_nsec;
}
